#include <hip/hip_runtime.h>

#include <vector>
#include <cstdio>
#include <cstdlib>

__global__ void kernel(int* x, int* y, int n) {
  // TODO: fill this in with the formula to calculate thread ID
  // size_t tid = ...;

  // TODO: fill in the guard condition
  /*
  if (...) {
    // x[:] += y[:] (add each element of y to the corresponding element of x,
    //               and store the result in x)
    ...;
  }
  */
}

int main(int argc, char** argv) {

  size_t n = 1000;

  std::vector<int> x(n, 1);
  std::vector<int> y(n, 1);

  int* d_x;
  hipMalloc(&d_x, sizeof(int)*n);

  int* d_y;
  hipMalloc(&d_y, sizeof(int)*n);

  hipMemcpy(d_x, x.data(), sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y.data(), sizeof(int)*n, hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  size_t block_size = 256;

  // ceil(grid_size / block_size)
  dim3 grid((n + block_size - 1) / block_size);
  dim3 block(block_size);

  kernel<<<grid, block>>>(d_x, d_y, n);

  hipMemcpy(x.data(), d_x, sizeof(int)*n, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  bool all_twos = true;
  for (size_t i = 0; i < x.size(); i++) {
    if (x[i] != 2) {
      all_twos = false;
      break;
    }
  }

  if (all_twos) {
    printf("OK!\n");
  } else {
    printf("FAILED.\n");
  }

  hipFree(d_x);
  hipFree(d_y);

  return 0;
}
