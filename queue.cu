#include <hip/hip_runtime.h>

#include <vector>
#include <cstdio>
#include <cstdlib>

template <typename T, std::size_t capacity>
struct queue {
  int size = 0;
  T data[capacity];

  __device__ bool insert(const T& value) {
    // TODO: insert an element into the queue.
    //       This will involve:
    //       1) An atomic increment to `size` using atomicAdd().
    //          Note that you can get a pointer to size with &size.
    //       2) If you've not overrun the end of the queue, write
    //          the element to the reserved slot in data.
  }
};

constexpr size_t queue_size = 1000;

__global__ void kernel(queue<int, queue_size>* queues, int n) {
  // TODO: have each thread insert its TID into every queue.
}

int main(int argc, char** argv) {

  constexpr size_t n = queue_size;

  std::vector<queue<int, n>> queues(n);

  queue<int, n>* d_queues;
  hipMalloc(&d_queues, sizeof(queue<int, n>)*n);

  hipMemcpy(d_queues, queues.data(), sizeof(queue<int, n>)*n, hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  size_t block_size = 256;

  // ceil(grid_size / block_size)
  dim3 grid((n + block_size - 1) / block_size);
  dim3 block(block_size);

  kernel<<<grid, block>>>(d_queues, n);

  hipMemcpy(queues.data(), d_queues, sizeof(queue<int, n>)*n, hipMemcpyDeviceToHost);

  bool success = true;

  for (size_t i = 0; i < n; i++) {
    queue<int, n>& queue = queues[i];

    if (queue.size != n) {
      success = false;
      break;
    }
    std::vector<size_t> histogram(n, 0);

    for (size_t i = 0; i < n; i++) {
      if (queue.data[i] < 0 && queue.data[i] >= n) {
        success = false;
        break;
      }
      histogram[queue.data[i]] += 1;
      if (histogram[queue.data[i]] != 1) {
        success = false;
        break;
      }
    }
  }

  if (success) {
    printf("OK!\n");
  } else {
    printf("FAILED.\n");
  }

  hipDeviceSynchronize();

  hipFree(d_queues);

  return 0;
}
