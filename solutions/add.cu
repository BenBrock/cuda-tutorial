#include <hip/hip_runtime.h>

#include <vector>
#include <cstdio>
#include <cstdlib>

__global__ void kernel(int* x, int* y, int n) {
  size_t tid = threadIdx.x + blockIdx.x*blockDim.x;

  if (tid < n) {
    x[tid] += y[tid];
  }
}

int main(int argc, char** argv) {

  size_t n = 1000;

  std::vector<int> x(n, 1);
  std::vector<int> y(n, 1);

  int* d_x;
  hipMalloc(&d_x, sizeof(int)*n);

  int* d_y;
  hipMalloc(&d_y, sizeof(int)*n);

  hipMemcpy(d_x, x.data(), sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y.data(), sizeof(int)*n, hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  size_t block_size = 256;

  // ceil(grid_size / block_size)
  dim3 grid((n + block_size - 1) / block_size);
  dim3 block(block_size);

  kernel<<<grid, block>>>(d_x, d_y, n);

  hipMemcpy(x.data(), d_x, sizeof(int)*n, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  bool all_twos = true;
  for (size_t i = 0; i < x.size(); i++) {
    if (x[i] != 2) {
      all_twos = false;
      break;
    }
  }

  if (all_twos) {
    printf("OK!\n");
  } else {
    printf("FAILED.\n");
  }

  hipFree(d_x);
  hipFree(d_y);

  return 0;
}
