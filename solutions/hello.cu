#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>

__global__ void kernel(size_t n_to_print) {
  size_t tid = threadIdx.x + blockIdx.x*blockDim.x;

  if (tid < n_to_print) {
    printf("Hello from thread %lu!\n", tid);
  }
}

int main(int argc, char** argv) {

  size_t grid_size = 1000;
  size_t block_size = 256;

  // ceil(grid_size / block_size)
  dim3 grid((grid_size + block_size - 1)/ block_size);
  dim3 block(block_size);

  kernel<<<grid, block>>>(grid_size);

  hipDeviceSynchronize();

  return 0;
}
