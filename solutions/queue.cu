#include <hip/hip_runtime.h>

#include <vector>
#include <cstdio>
#include <cstdlib>

template <typename T, std::size_t capacity>
struct queue {
  int size = 0;
  T data[capacity];

  __device__ bool insert(const T& value) {
    int result = atomicAdd(&size, 1);
    if (result >= capacity) {
      // Queue is overflowing.  Do nothing.
      return false;
    } else {
      data[result] = value;
      return true;
    }
  }
};

constexpr size_t queue_size = 1000;

__global__ void kernel(queue<int, queue_size>* queues, int n) {
  size_t tid = threadIdx.x + blockIdx.x*blockDim.x;

  if (tid < n) {
    for (size_t i = 0; i < n; i++) {
      queues[i].insert(tid);
    }
  }
}

int main(int argc, char** argv) {

  constexpr size_t n = queue_size;

  std::vector<queue<int, n>> queues(n);

  queue<int, n>* d_queues;
  hipMalloc(&d_queues, sizeof(queue<int, n>)*n);

  hipMemcpy(d_queues, queues.data(), sizeof(queue<int, n>)*n, hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  size_t block_size = 256;

  // ceil(grid_size / block_size)
  dim3 grid((n + block_size - 1) / block_size);
  dim3 block(block_size);

  kernel<<<grid, block>>>(d_queues, n);

  hipMemcpy(queues.data(), d_queues, sizeof(queue<int, n>)*n, hipMemcpyDeviceToHost);

  bool success = true;

  for (size_t i = 0; i < n; i++) {
    queue<int, n>& queue = queues[i];

    if (queue.size != n) {
      success = false;
      break;
    }
    std::vector<size_t> histogram(n, 0);

    for (size_t i = 0; i < n; i++) {
      if (queue.data[i] < 0 && queue.data[i] >= n) {
        success = false;
        break;
      }
      histogram[queue.data[i]] += 1;
      if (histogram[queue.data[i]] != 1) {
        success = false;
        break;
      }
    }
  }

  if (success) {
    printf("OK!\n");
  } else {
    printf("FAILED.\n");
  }

  hipDeviceSynchronize();

  hipFree(d_queues);

  return 0;
}
