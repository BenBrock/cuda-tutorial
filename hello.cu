#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>

__global__ void kernel(size_t n_to_print) {
  // TODO: fill this in with the formula to calculate thread ID
  // size_t tid = ...;

  // TODO: fill in the parameter for this *guard*
  /*
  if (...) {
    printf("Hello from thread %lu!\n", tid);
  }
  */
}

int main(int argc, char** argv) {

  size_t grid_size = 1000;
  size_t block_size = 256;

  // ceil(grid_size / block_size)
  dim3 grid((grid_size + block_size - 1)/ block_size);
  dim3 block(block_size);

  kernel<<<grid, block>>>(grid_size);

  hipDeviceSynchronize();

  return 0;
}
