#include <hip/hip_runtime.h>

#include <vector>
#include <cstdio>
#include <cstdlib>

__global__ void kernel(int* x, int* y, int m, int n) {
  // TODO: add formulas for TID in the x and y dimensions.
  // size_t tidx = ...;
  // size_t tidy = ...;

  // TODO: you'll need a guard in two dimensions.
  /*
  if (...) {
    // TODO: Set each element (tidx, tidy) x[tidx,tidy] += y[tidx,tidy]
    x[tidx*n + tidy] += y[tidx*n + tidy];
  }
  */
}

int main(int argc, char** argv) {

  size_t m = 1000;
  size_t n = 1000;

  std::vector<int> x(m*n, 1);
  std::vector<int> y(m*n, 1);

  int* d_x;
  hipMalloc(&d_x, sizeof(int)*m*n);

  int* d_y;
  hipMalloc(&d_y, sizeof(int)*m*n);

  hipMemcpy(d_x, x.data(), sizeof(int)*m*n, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y.data(), sizeof(int)*m*n, hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  size_t block_size = 16;

  // ceil(grid_size / block_size)
  dim3 grid((m + block_size - 1) / block_size,
            (n + block_size - 1) / block_size);
  dim3 block(block_size, block_size);

  kernel<<<grid, block>>>(d_x, d_y, m, n);

  hipMemcpy(x.data(), d_x, sizeof(int)*m*n, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  bool all_twos = true;
  for (size_t i = 0; i < x.size(); i++) {
    if (x[i] != 2) {
      printf("Breaking with %lu == %d\n", i, x[i]);
      all_twos = false;
      break;
    }
  }

  if (all_twos) {
    printf("OK!\n");
  } else {
    printf("FAILED.\n");
  }

  hipFree(d_x);
  hipFree(d_y);

  return 0;
}
